#include "hip/hip_runtime.h"
//#include "cuda_source.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""
#include "error_utils.h"
#include <hip/hip_complex.h>
#include <complex.h>

inline hipDoubleComplex operator + (hipDoubleComplex c1, hipDoubleComplex c2) { return hipCadd(c1, c2); }
inline hipDoubleComplex operator - (hipDoubleComplex c1, hipDoubleComplex c2) { return hipCsub(c1, c2); }
   
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void vectorAddComplex(const hipDoubleComplex* A, const hipDoubleComplex* B, hipDoubleComplex* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}



/**
 * Host main routine
 */
extern "C" {
    _Dcomplex* vector_add_gpu_complex(_Dcomplex* h_A_input, _Dcomplex* h_B_input, int numElements) {

        hipDoubleComplex* h_A = (hipDoubleComplex*)h_A_input;
        hipDoubleComplex* h_B = (hipDoubleComplex*)h_B_input;
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        size_t size = numElements * sizeof(hipDoubleComplex);

        printf("[Vector addition of %d (complex) elements]\n", numElements);

        // Allocate the host output vector C
        hipDoubleComplex* h_C = (hipDoubleComplex*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate complex host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector A
        hipDoubleComplex* d_A = NULL;
        err = hipMalloc((void**)&d_A, size);
        checkErr(err, "Allocating device vector A");

        // Allocate the device input vector B
        hipDoubleComplex* d_B = NULL;
        err = hipMalloc((void**)&d_B, size);
        checkErr(err, "Allocate device vector B");

        // Allocate the device output vector C
        hipDoubleComplex* d_C = NULL;
        err = hipMalloc((void**)&d_C, size);
        checkErr(err, "Allocate device vector C");

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy A host to device");

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy B host to device");

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        vectorAddComplex << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
        err = hipGetLastError();
        checkErr(err, "Launch vectorAdd kernel");

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        checkErr(err, "Copy C device to host");

        // Verify that the result vector is correct
        for (int i = 0; i < numElements; ++i)
        {
            if (hipCabs(hipCsub(hipCadd(h_A[i],h_B[i]), h_C[i])) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        printf("Test PASSED\n");

        // Free device global memory
        err = hipFree(d_A);
        checkErr(err, "Free device vector A");

        err = hipFree(d_B);
        checkErr(err, "Free device vector B");

        err = hipFree(d_C);
        checkErr(err, "Free device vector C");

        printf("Done\n");
        return (_Dcomplex*) h_C;
    }

    double* vector_add_gpu(double* h_A, double* h_B, int numElements) {

        _Dcomplex *a = new _Dcomplex { 1.0,2.0 };
        hipDoubleComplex *c = (hipDoubleComplex* )a;
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;
        double cr = hipCreal(*c);
        double ci = hipCimag(*c);
        printf("r,i: %f,%f\n", cr, ci);

        size_t size = numElements * sizeof(double);

        printf("[Vector addition of %d elements]\n", numElements);

        // Allocate the host output vector C
        double* h_C = (double*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector A
        double* d_A = NULL;
        err = hipMalloc((void**)&d_A, size);
        checkErr(err, "Allocating device vector A");

        // Allocate the device input vector B
        double* d_B = NULL;
        err = hipMalloc((void**)&d_B, size);
        checkErr(err, "Allocate device vector B");

        // Allocate the device output vector C
        double* d_C = NULL;
        err = hipMalloc((void**)&d_C, size);
        checkErr(err, "Allocate device vector C");

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy A host to device");

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy B host to device");

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
        err = hipGetLastError();
        checkErr(err, "Launch vectorAdd kernel");

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        checkErr(err, "Copy C device to host");

        // Verify that the result vector is correct
        for (int i = 0; i < numElements; ++i)
        {
            if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        printf("Test PASSED\n");

        // Free device global memory
        err = hipFree(d_A);
        checkErr(err, "Free device vector A");

        err = hipFree(d_B);
        checkErr(err, "Free device vector B");

        err = hipFree(d_C);
        checkErr(err, "Free device vector C");

        printf("Done\n");
        return h_C;
    }
}