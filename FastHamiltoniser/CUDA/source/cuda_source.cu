#include "hip/hip_runtime.h"
//#include "cuda_source.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""
#include "error_utils.h"
#include <hip/hip_complex.h>
#include <complex.h>
#include "..\..\HamiltonianMatrix.h"

__device__ inline hipDoubleComplex operator + (hipDoubleComplex c1, hipDoubleComplex c2) { return hipCadd(c1, c2); }
__device__ inline hipDoubleComplex operator - (hipDoubleComplex c1, hipDoubleComplex c2) { return hipCsub(c1, c2); }
__device__ inline hipDoubleComplex operator * (hipDoubleComplex c1, hipDoubleComplex c2) { return hipCmul(c1, c2); }
   
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void vectorAddComplex(const hipDoubleComplex* A, const hipDoubleComplex* B, hipDoubleComplex* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void permuteKernel(int permute_size, const int* targets, const int* sources, const hipDoubleComplex* relative_couplings,
    hipDoubleComplex coefficient, hipDoubleComplex* output, const hipDoubleComplex* input)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < permute_size)
    {
        output[targets[i]] = output[targets[i]] + (coefficient * input[sources[i]] * relative_couplings[i]);
    }
}



/**
 * Host main routine
 */
extern "C" {

    void apply_hamiltonian_gpu(HamiltonianMatrix* ham, _Dcomplex* h_input, _Dcomplex* h_output) {
        //On my GPU (p620) 4 streaming multiprocessors; each with upto 2048 threads that can be divided in up to 32 thread blocks -> 64 threads per block.
        //hipDoubleComplex* h_output = (hipDoubleComplex*) malloc(ham->dim*sizeof(hipDoubleComplex));
        //hipDoubleComplex* h_input = (hipDoubleComplex*)output_vector;
        hipError_t err = hipSuccess;

        size_t size = ham->dim * sizeof(hipDoubleComplex);

        hipDoubleComplex* d_input = NULL;
        err = hipMalloc((void**)&d_input, size);
        checkErr(err, "Allocating device vector d_input");

        hipDoubleComplex* d_output = NULL;
        err = hipMalloc((void**)&d_output, size);
        checkErr(err, "Allocating device vector d_output");

        err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy input host to device");

        err = hipMemcpy(d_output, h_output, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy output host to device");

        void** to_free = (void**)malloc(sizeof(void*) * ham->n_permuters*3);

        for (int p_i = 0; p_i < ham->n_permuters; p_i++) {
            Permuter* permuter = ham->permuters + p_i;
            if (!permuter->active) continue;
            hipDoubleComplex coefficient = *(hipDoubleComplex*)&permuter->coefficient;
            
            size_t int_array_size = permuter->size * sizeof(int);
            size_t coeff_array_size = permuter->size * sizeof(hipDoubleComplex);

            //Allocating - also keep track of pointers to free at the end
            int* d_targets = NULL;
            err = hipMalloc((void**)&d_targets, int_array_size);
            checkErr(err, "Allocating device vector d_targets");
            to_free[3*p_i] = d_targets;

            int* d_sources = NULL;
            err = hipMalloc((void**)&d_sources, int_array_size);
            checkErr(err, "Allocating device vector d_sources");
            to_free[3*p_i+1] = d_sources;

            hipDoubleComplex* d_relative_coeff = NULL;
            err = hipMalloc((void**)&d_relative_coeff, coeff_array_size);
            checkErr(err, "Allocating device vector d_relative_coeff");
            to_free[3*p_i + 2] = d_relative_coeff;

            //Copying
            err = hipMemcpy(d_targets, permuter->targets, int_array_size, hipMemcpyHostToDevice);
            checkErr(err, "Copy targets host to device");

            err = hipMemcpy(d_sources, permuter->sources, int_array_size, hipMemcpyHostToDevice);
            checkErr(err, "Copy sources host to device");

            err = hipMemcpy(d_relative_coeff, permuter->relative_couplings, coeff_array_size, hipMemcpyHostToDevice);
            checkErr(err, "Copy relative_coefficients host to device");

            

            int threadsPerBlock = 64;
            int blocksPerGrid = (permuter->size + threadsPerBlock - 1) / threadsPerBlock;
            printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
            permuteKernel << <blocksPerGrid, threadsPerBlock >> > (
                permuter->size,d_targets,d_sources,d_relative_coeff,coefficient, d_output,d_input);
            err = hipGetLastError();
            checkErr(err, "Launch vectorAdd kernel");

        }
        err = hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
        checkErr(err, "Copy C device to host");


        //Free memory
        err = hipFree(d_input);
        checkErr(err, "Free device vector d_input");
        err = hipFree(d_output);
        checkErr(err, "Free device vector d_output");
        for (int p_i = 0; p_i < ham->n_permuters; p_i++) { 
            if (ham->permuters[p_i].active) {
                err = hipFree(to_free[3*p_i]);
                checkErr(err, "Free permuter targets.");
                err = hipFree(to_free[3*p_i + 1]);
                checkErr(err, "Free permuter sources.");
                err = hipFree(to_free[3*p_i + 2]);
                checkErr(err, "Free permuter relative_coeff.");
            }
        }
        free(to_free);
    }

    _Dcomplex* make_identity(int dim) {
        return NULL;
    }
    _Dcomplex* vector_add_gpu_complex(_Dcomplex* h_A_input, _Dcomplex* h_B_input, int numElements) {

        hipDoubleComplex* h_A = (hipDoubleComplex*)h_A_input;
        hipDoubleComplex* h_B = (hipDoubleComplex*)h_B_input;
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;

        size_t size = numElements * sizeof(hipDoubleComplex);

        printf("[Vector addition of %d (complex) elements]\n", numElements);

        // Allocate the host output vector C
        hipDoubleComplex* h_C = (hipDoubleComplex*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate complex host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector A
        hipDoubleComplex* d_A = NULL;
        err = hipMalloc((void**)&d_A, size);
        checkErr(err, "Allocating device vector A");

        // Allocate the device input vector B
        hipDoubleComplex* d_B = NULL;
        err = hipMalloc((void**)&d_B, size);
        checkErr(err, "Allocate device vector B");

        // Allocate the device output vector C
        hipDoubleComplex* d_C = NULL;
        err = hipMalloc((void**)&d_C, size);
        checkErr(err, "Allocate device vector C");


        printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy A host to device");

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy B host to device");

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        vectorAddComplex << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
        err = hipGetLastError();
        checkErr(err, "Launch vectorAdd kernel");

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        checkErr(err, "Copy C device to host");

        // Verify that the result vector is correct
        for (int i = 0; i < numElements; ++i)
        {
            if (hipCabs(hipCsub(hipCadd(h_A[i],h_B[i]), h_C[i])) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        printf("Test PASSED\n");

        // Free device global memory
        err = hipFree(d_A);
        checkErr(err, "Free device vector A");

        err = hipFree(d_B);
        checkErr(err, "Free device vector B");

        err = hipFree(d_C);
        checkErr(err, "Free device vector C");

        printf("Done\n");
        return (_Dcomplex*) h_C;
    }

    double* vector_add_gpu(double* h_A, double* h_B, int numElements) {

        _Dcomplex *a = new _Dcomplex { 1.0,2.0 };
        hipDoubleComplex *c = (hipDoubleComplex* )a;
        // Error code to check return values for CUDA calls
        hipError_t err = hipSuccess;
        double cr = hipCreal(*c);
        double ci = hipCimag(*c);
        printf("r,i: %f,%f\n", cr, ci);

        size_t size = numElements * sizeof(double);

        printf("[Vector addition of %d elements]\n", numElements);

        // Allocate the host output vector C
        double* h_C = (double*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector A
        double* d_A = NULL;
        err = hipMalloc((void**)&d_A, size);
        checkErr(err, "Allocating device vector A");

        // Allocate the device input vector B
        double* d_B = NULL;
        err = hipMalloc((void**)&d_B, size);
        checkErr(err, "Allocate device vector B");

        // Allocate the device output vector C
        double* d_C = NULL;
        err = hipMalloc((void**)&d_C, size);
        checkErr(err, "Allocate device vector C");

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy A host to device");

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        checkErr(err, "Copy B host to device");

        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
        err = hipGetLastError();
        checkErr(err, "Launch vectorAdd kernel");

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        printf("Copy output data from the CUDA device to the host memory\n");
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        checkErr(err, "Copy C device to host");

        // Verify that the result vector is correct
        for (int i = 0; i < numElements; ++i)
        {
            if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        printf("Test PASSED\n");

        // Free device global memory
        err = hipFree(d_A);
        checkErr(err, "Free device vector A");

        err = hipFree(d_B);
        checkErr(err, "Free device vector B");

        err = hipFree(d_C);
        checkErr(err, "Free device vector C");

        printf("Done\n");
        return h_C;
    }
}